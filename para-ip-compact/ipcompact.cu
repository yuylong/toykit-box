#include "hip/hip_runtime.h"
/*-
 * GNU GENERAL PUBLIC LICENSE, version 3
 * See LICENSE file for detail.
 *
 * Author: Yulong Yu
 * Copyright(c) 2018 Yulong Yu. All rights reserved.
 */

#include <stdio.h>
#include <stdlib.h>

#include "ipcompact.h"

#define CUDA_BLOCK_SIZE    8

#define IP_STR_MAXLEN     16
#define IP_CONTI_THRES     2

__global__ void ip_str_to_bin (char ipstr[], int iplist[], int num)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx >= num )
        return;
        
    int ipbin = 0, ipsect = 0;
    char *p = &ipstr[idx * IP_STR_MAXLEN];
    
    // Iterate. Any unvisible char will terminate the process.
    while ( *p >= 32 && *p <= 127 ) {
        if ( *p == '.' ) {
            ipbin = ipbin << 8 + ipsect;
            ipsect = 0;
        } else if ( *p >= '0' && *p <= '9' ) {
            ipsect = ipsect * 10 + *p - '0';
        } else {
            ipbin = ipsect = 0;
            break;
        }
        p++;
    }
    ipbin = ipbin << 8 + ipsect;
    iplist[idx] = ipbin;
}

__global__ void ip_conti_label (unsigned int iplist[], unsigned int label[], int num)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int step = 1;
    __shared__ unsigned int shd_label[CUDA_BLOCK_SIZE];
    __shared__ bool shd_dead[CUDA_BLOCK_SIZE];
    //extern __shared__ unsigned int shd_label[/*CUDA_BLOCK_SIZE * 2*/];
    //bool *shd_dead = (bool *)(shd_label + CUDA_BLOCK_SIZE);

    if ( idx >= num )
        return;

    if ( iplist[idx + 1] - iplist[idx] <= IP_CONTI_THRES && idx < num - 1 ) {
        shd_label[threadIdx.x] = 1;
        if ( threadIdx.x == CUDA_BLOCK_SIZE - 1 )
            shd_dead[threadIdx.x] = true;
        else
            shd_dead[threadIdx.x] = false;
    } else {
        shd_label[threadIdx.x] = 0;
        shd_dead[threadIdx.x] = true;
    }

    __syncthreads ();

    while ( !shd_dead[threadIdx.x] ) {
       shd_label[threadIdx.x] += shd_label[threadIdx.x + step];
       shd_dead[threadIdx.x] = shd_dead[threadIdx.x + step];

       step *= 2;
       __syncthreads ();
    }

    label[idx] = shd_label[threadIdx.x];
}

__global__ void ip_conti_label_finish (unsigned int label[], int blocknum, int blocksize)
{
    if ( blockIdx.x > 0 )
        return;
    int blkidx = threadIdx.x;
    
    while ( blkidx < blocknum - 1 ) {
        int idx = (blkidx + 1) * blocksize;
        int valf = label[idx];
        if ( valf == blocksize )
            goto iter_skip;
        
        while ( idx > 0 ) {
            if ( label[idx - blocksize] == blocksize ) {
                valf += blocksize;
                idx -= blocksize;
            } else {
                break;
            }
        }

        if ( label[idx - 1] == 0 ) {
            label[idx] = valf;
            goto iter_skip;
        }
        
        idx--;
        while ( idx > 0 ) {
            if ( label[idx - 1] == 0 )
                break;
            idx--;
        }
        label[idx] += valf;
        
    iter_skip:
        blkidx += blockDim.x;
    }
}

__global__ void ip_conti_gather (unsigned int label[], unsigned int glbl_pos[], unsigned int glbl_siz[], int blk_gther_size[], int num)
{
    int blkoff = blockIdx.x * blockDim.x;
    int idx = blkoff + threadIdx.x;
    __shared__ int shd_gth_idx[1];

    if ( threadIdx.x == 0 )
        shd_gth_idx[0] = 0;
    __syncthreads();

    if ( idx >= num )
        return;

    if ( idx == 0 || label[idx - 1] == 0 ) {
        int gth_idx = atomicAdd (&shd_gth_idx[0], 1);
        glbl_pos[blkoff + gth_idx] = idx;
        glbl_siz[blkoff + gth_idx] = label[idx] + 1;
    }

    if ( threadIdx.x != 0 )
        return;
    blk_gther_size[blockIdx.x] = shd_gth_idx[0];
}

__global__ void ip_conti_gather_idxscan (int blk_gther_size[], int out_blk_gther_size[], int blknum)
{
    if ( blockIdx.x > 0 )
        return;

    int idx = threadIdx.x, step = 1;
    int gtherbase = 0;
    __shared__ int shdarray[CUDA_BLOCK_SIZE];
    
    while ( idx < blknum ) {
        shdarray[threadIdx.x] = blk_gther_size[idx];
        __syncthreads();
        
        printf("tid=%u, idx=%u, AAA\n", threadIdx.x, idx);
        
        step = 1;
        while ( step < CUDA_BLOCK_SIZE ) {
            if ( threadIdx.x >= step )
                shdarray[threadIdx.x] += shdarray[threadIdx.x - step];
            step *= 2;
            __syncthreads();
        }
        
        out_blk_gther_size[idx] = shdarray[threadIdx.x] + gtherbase;
        if ( threadIdx.x == blockDim.x - 1 )
            gtherbase += shdarray[threadIdx.x];
            
        idx += blockDim.x;
    }
}

__global__ void ip_conti_gather_finish (unsigned int iplist[],
                                        unsigned int glbl_pos[], unsigned int glbl_siz[], int blk_gther_size[], 
                                        struct ipnode outlist[])
{
    __shared__ int localinfo[2];
    if ( threadIdx.x == 0 ) {
        localinfo[0] = ( blockIdx.x >= 1 ? blk_gther_size[blockIdx.x - 1] : 0 );
        localinfo[1] = blk_gther_size[blockIdx.x] - localinfo[0];
    }
    __syncthreads();
    
    if ( threadIdx.x >= localinfo[1] )
        return;

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int outidx = localinfo[0] + threadIdx.x;
    int sidx = glbl_pos[idx];
    int eidx = sidx + glbl_siz[idx] - 1;
    
    outlist[outidx].ip = iplist[sidx];
    if ( eidx == sidx ) {
        outlist[outidx].type = IPNODE_TYPE_SINGLE;
    } else {
        outlist[outidx].type = IPNODE_TYPE_SCOPE;
        outlist[outidx].u.end_ip = iplist[eidx];
    }
}


int ip_compact (unsigned int iplist[], int num,
                struct ipnode outlist[], int *outnum)
{
    unsigned int *dev_iplist;
    unsigned int *dev_label;
    unsigned int *dev_gath_label, *dev_glbl_pos, *dev_glbl_siz;
    int *dev_blk_gther_size, *dev_out_blk_gther_size;
    struct ipnode *dev_outlist;
    int gridsize, blocksize;

    unsigned int *hst_label;
    unsigned int *hst_gath_label, *hst_glbl_pos, *hst_glbl_siz;
    int *hst_blk_gther_size, *hst_out_blk_gther_size;
    int i, j;

    hipMalloc ((void **)&dev_iplist, sizeof (unsigned int) * num);
    hipMalloc ((void **)&dev_label, sizeof (unsigned int) * num);
    hipMalloc ((void **)&dev_glbl_pos, sizeof (unsigned int) * num);
    hipMalloc ((void **)&dev_glbl_siz, sizeof (unsigned int) * num);
    hipMalloc ((void **)&dev_outlist, sizeof (struct ipnode) * num);

    hipMemcpy (dev_iplist, iplist, sizeof (unsigned int) * num,
                hipMemcpyHostToDevice);

    blocksize = CUDA_BLOCK_SIZE;
    gridsize = (num + blocksize - 1) / blocksize;
    hipMalloc ((void **)&dev_blk_gther_size, sizeof (int) * gridsize);
    hipMalloc ((void **)&dev_out_blk_gther_size, sizeof (int) * gridsize);

    ip_conti_label <<<gridsize, blocksize>>> (dev_iplist, dev_label, num);
    if ( gridsize > 1 )
        ip_conti_label_finish <<<1, blocksize>>> (dev_label, gridsize, blocksize);
    ip_conti_gather <<<gridsize, blocksize>>> (dev_label, dev_glbl_pos, dev_glbl_siz, dev_blk_gther_size, num);
    ip_conti_gather_idxscan <<<1, blocksize>>> (dev_blk_gther_size, dev_out_blk_gther_size, gridsize);
    ip_conti_gather_finish <<<gridsize, blocksize>>> (dev_iplist, dev_glbl_pos, dev_glbl_siz, dev_out_blk_gther_size, dev_outlist);

    hst_label = (unsigned int *)malloc (sizeof (unsigned int) * num);
    hipMemcpy (hst_label, dev_label, sizeof (unsigned int) * num,
                hipMemcpyDeviceToHost);

    
    hst_glbl_pos = (unsigned int *)malloc (sizeof (unsigned int) * num);
    hst_glbl_siz = (unsigned int *)malloc (sizeof (unsigned int) * num);
    hipMemcpy (hst_glbl_pos, dev_glbl_pos, sizeof (unsigned int) * num,
                hipMemcpyDeviceToHost);
    hipMemcpy (hst_glbl_siz, dev_glbl_siz, sizeof (unsigned int) * num,
                hipMemcpyDeviceToHost);

    hst_blk_gther_size = (int *)malloc (sizeof (int) * gridsize);
    hst_out_blk_gther_size = (int *)malloc (sizeof (int) * gridsize);
    hipMemcpy (hst_blk_gther_size, dev_blk_gther_size, sizeof (int) * gridsize,
                hipMemcpyDeviceToHost);
    hipMemcpy (hst_out_blk_gther_size, dev_out_blk_gther_size, sizeof (int) * gridsize,
                hipMemcpyDeviceToHost);
                
    hipMemcpy (outlist, dev_outlist, sizeof (struct ipnode) * num,
                hipMemcpyDeviceToHost);

    for ( i = 0; i < num; i++ ) {
        printf ("%s%d", (i == 0 ? "" : ", "), hst_label[i]);
    }
    printf ("\n");

    printf ("Gathered Info:\n");
    for ( i = 0; i < gridsize; i++ ) {
        printf("Block-%d (%d/%d): ", i, hst_blk_gther_size[i], hst_out_blk_gther_size[i]);
        int blkoff = i * blocksize;
        for ( j = 0; j < hst_blk_gther_size[i]; j++ ) {
            printf("%s%u@%u", (j == 0 ? "" : ", "), hst_glbl_siz[blkoff + j] , hst_glbl_pos[blkoff + j]);
        }
        printf("\n");
    }

    printf ("Result:\n");
    for ( i = 0; i < hst_out_blk_gther_size[gridsize - 1]; i++ ) {
        printf ("%s", (i == 0 ? "" : ", "));
        
        if ( outlist[i].type == IPNODE_TYPE_DUMMY )
            printf ("ERROR");
        else
            printf ("%08X", outlist[i].ip);
        
        if ( outlist[i].type == IPNODE_TYPE_SCOPE)
            printf ("--%08X", outlist[i].u.end_ip);
        else if ( outlist[i].type == IPNODE_TYPE_NET)
            printf ("/%u", outlist[i].u.masklen);
    }
    printf ("\n");

    free (hst_label);
    free (hst_glbl_pos);
    free (hst_glbl_siz);
    free (hst_blk_gther_size);
    free (hst_out_blk_gther_size);

    return 0;
}
